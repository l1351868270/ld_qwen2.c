#include "hip/hip_runtime.h"
/*
make qwen2
python run.py --model_type=Qwen/Qwen1.5-0.5B-Chat --prompt="天空为什么是蓝色的,答案大于1000字"
*/

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <math.h>
#include <fcntl.h>
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <unistd.h>
#include <sys/mman.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include "./src/kernels/cuda/embedding.cuh"
#include "./src/kernels/cuda/rope_half.cuh"
#include "./src/kernels/cuda/rmsnorm_half.cuh"
#include "./src/kernels/cuda/linear_half.cuh"
#include "./src/kernels/cuda/flash_attention_half.cuh"
#include "./src/kernels/cuda/residual_half.cuh"
#include "./src/kernels/cuda/silu_half.cuh"
#include "./src/kernels/cuda/argmax_half.cuh"

extern "C" {
    void c_init(int batch, int max_seq_len, const char *checkpoint_path);
    int* c_qwen2_forward(int batch, int seq_len, int *data, int pos);
    // void c_generate(int batch, int seq_len, int *data, int steps);
    // void c_chat ();
}

constexpr int MODEL_LIANMENT{16};

typedef struct {
    int hidden_size;
    int intermediate_size;
    int max_position_embeddings;
    int max_window_layers;
    int num_attention_heads;
    int num_hidden_layers;
    int num_key_value_heads;
    float rms_norm_eps;
    float rope_theta;
    int sliding_window;
    int vocab_size;
} Qwen2Config;

typedef struct {
    half *embed_tokens;    // model.embed_tokens.weight
    half *q_proj_w;        // model.layers.{i}.self_attn.q_proj.weight
    half *q_proj_b;        // model.layers.{i}.self_attn.q_proj.bias
    half *k_proj_w;        // model.layers.{i}.self_attn.k_proj.weight
    half *k_proj_b;        // model.layers.{i}.self_attn.k_proj.bias
    half *v_proj_w;        // model.layers.{i}.self_attn.v_proj.weight
    half *v_proj_b;        // model.layers.{i}.self_attn.v_proj.bias
    half *o_proj;          // model.layers.{i}.self_attn.o_proj.weight
    half *gate_proj;       // model.layers.{i}.mlp.gate_proj.weight
    half *up_proj;                    // model.layers.{i}.mlp.up_proj.weight
    half *down_proj;                  // model.layers.{i}.mlp.down_proj.weight
    half *input_layernorm;            // model.layers.{i}.input_layernorm.weight
    half *post_attention_layernorm;   // model.layers.{i}.post_attention_layernorm.weight
    half *norm;            // model.norm.weight
    half *lm_head;         // lm_head.weight
} Qwen2Weights;

typedef struct {
    half *half_x;
    half *half_xb;
    half *half_xb2;
    half *half_hb;
    half *half_hb2;
    half *half_q;
    half *half_k;
    half *half_v;
    half *half_key_cache;
    half *half_value_cache;
    half *half_logits;
    int *next;
    int *token;
    int *next_cpu;
    int1 a;
    int batch;
    int max_seq_len;

    int flops;
    int flops_sfu;

    int num_parameters;
    hipblasHandle_t *handle;
} RunState;

typedef struct {
    Qwen2Config config;
    RunState state;
    Qwen2Weights weights;
} Qwen2;

void malloc_run_state(RunState* s, Qwen2Config* p) {
    int seq_len = s->max_seq_len;
    int batch = s->batch;
    int hidden_size = p->hidden_size;
    int intermediate_size = p->intermediate_size;
    
    int num_heads = p->num_attention_heads;
    int head_dim = p->hidden_size / num_heads;
    int num_key_value_heads = p->num_key_value_heads;

    int num_hidden_layers = p->num_hidden_layers;

    unsigned long long run_cache = 0;

    hipMalloc((void**)&s->half_x, batch * hidden_size * sizeof(half));
    run_cache += batch * hidden_size * sizeof(half);
    hipMalloc((void**)&s->half_xb, batch * hidden_size * sizeof(half));
    run_cache += batch * hidden_size * sizeof(half);
    hipMalloc((void**)&s->half_xb2, batch * hidden_size * sizeof(half));
    run_cache += batch * hidden_size * sizeof(half);
    hipMalloc((void**)&s->half_hb, batch * intermediate_size * sizeof(half));
    run_cache += batch * intermediate_size * sizeof(half);
    hipMalloc((void**)&s->half_hb2, batch * intermediate_size * sizeof(half));
    run_cache += batch * intermediate_size * sizeof(half);
    hipMalloc((void**)&s->half_q, batch * hidden_size * sizeof(half));
    run_cache += batch * hidden_size * sizeof(half);
    unsigned long long kv_cache_size = batch * num_hidden_layers * seq_len * num_key_value_heads * head_dim * sizeof(half);
    hipMalloc((void**)&s->half_key_cache, kv_cache_size);
    run_cache += kv_cache_size;
    hipMalloc((void**)&s->half_value_cache, kv_cache_size);
    run_cache += kv_cache_size;
    printf("total kv cache size: %llu bytes, via %fKB, via %fMB, via %fGB\n", 2 * kv_cache_size, 
            (float)kv_cache_size  * 2.0 / 1024, (float)kv_cache_size  * 2.0 / 1024 / 1024, (float)kv_cache_size  * 2.0 / 1024 / 1024 / 1024);

    hipMalloc((void**)&s->half_logits, batch * p->vocab_size * sizeof(half));
    run_cache += batch * p->vocab_size * sizeof(half);

    hipMalloc((void**)&s->next, batch * sizeof(int));
    run_cache += batch * sizeof(int);
    hipMalloc((void**)&s->token, batch * sizeof(int));
    run_cache += batch * sizeof(int);
    printf("total run cache size: %llu bytes, via %fKB, via %fMB, via %fGB\n", run_cache, 
            (float)run_cache / 1024, (float)run_cache / 1024 / 1024, (float)run_cache / 1024 / 1024 / 1024);

    s->next_cpu = (int*)malloc(batch * sizeof(int));
}

void free_run_state(RunState* s) {
    hipFree(s->half_x);
    hipFree(s->half_xb);
    hipFree(s->half_xb2);
    hipFree(s->half_hb);
    hipFree(s->half_hb2);
    hipFree(s->half_q);
    hipFree(s->half_k);
    hipFree(s->half_v);
    hipFree(s->half_key_cache);
    hipFree(s->half_value_cache);
    hipFree(s->half_logits);
    free(s->next);
    free(s->token);
    free(s->next_cpu);
    hipFree(s->half_logits);
    hipFree(s->next);
    hipFree(s->token);
    free(s->next_cpu);
}

void parse_ll(char** ptr, unsigned long long *ll, unsigned long long *ll_bytes, char *weight_name = nullptr) {
    hipMemcpy(ll, *ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    *ptr += sizeof(unsigned long long);
    hipMemcpy(ll_bytes, *ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    *ptr += sizeof(unsigned long long);
#ifdef WEIGHTS_DEBUG
    printf("weights length is:       %llu\n", *ll);
    printf("weights bytes length is: %llu\n", *ll_bytes);
#endif
}

void memory_map_weights(Qwen2Weights *w, Qwen2Config* p, char* ptr) {
    unsigned long long ll;
    unsigned long long ll_bytes;;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->embed_tokens = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->q_proj_w = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->q_proj_b = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->k_proj_w = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->k_proj_b = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->v_proj_w = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->v_proj_b = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->o_proj = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->gate_proj = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->up_proj = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->down_proj = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->input_layernorm = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->post_attention_layernorm = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->norm = (half*)ptr;
    ptr += ll_bytes;
    parse_ll(&ptr, &ll, &ll_bytes);
    w->lm_head = (half*)ptr;
}

void qwen2_build_from_checkpoint(Qwen2 *model, const char* checkpoint_path) {
    FILE *model_file = fopen(checkpoint_path, "rb");
    if (model_file == NULL) {
        printf("Error opening model file %s\n", checkpoint_path);
    }
    char model_alignment[MODEL_LIANMENT];
    size_t file_size = 0;
    fseek(model_file, 0, SEEK_END);
    file_size = ftell(model_file);
    fseek(model_file, 0, SEEK_SET);

#ifdef WEIGHTS_DEBUG
    printf("file_size is: %ld\n", file_size);
#endif

    int rcount = 0;
    int model_magic;
    rcount = fread(&model_magic, sizeof(int), 1, model_file);
    if (rcount != 1) {
        fprintf(stderr, "Bad read magic from model file %s\n", checkpoint_path);
        exit(1);
    }
    
    if (model_magic != 20240516) {
        fprintf(stderr, "Bad magic model file %s\n", checkpoint_path);
        exit(1);
    }

#ifdef WEIGHTS_DEBUG
    printf("model magic is: %d\n", model_magic);
#endif
    rcount = fread(&model->config, sizeof(int), sizeof(model->config) / sizeof(int), model_file);
    if (rcount != sizeof(model->config) / sizeof(int)) {
        fprintf(stderr, "Bad read config from model file %s\n", checkpoint_path);
        exit(1);
    }

#ifdef WEIGHTS_DEBUG
    printf("config hidden_size is: %d\n", model->config.hidden_size);
    printf("config intermediate_size is: %d\n", model->config.intermediate_size);
    printf("config max_position_embeddings is: %d\n", model->config.max_position_embeddings);
    printf("config max_window_layers is: %d\n", model->config.max_window_layers);
    printf("config num_attention_heads is: %d\n", model->config.num_attention_heads);
    printf("config num_hidden_layers is: %d\n", model->config.num_hidden_layers);
    printf("config num_key_value_heads is: %d\n", model->config.num_key_value_heads);
    printf("config rms_norm_eps is: %f\n", model->config.rms_norm_eps);
    printf("config rope_theta is: %f\n", model->config.rope_theta);
    printf("config sliding_window is: %d\n", model->config.sliding_window);
    printf("config vocab_size is: %d\n", model->config.vocab_size);
#endif

    size_t head_bytes = sizeof(model->config) + sizeof(int);
    if (head_bytes % MODEL_LIANMENT != 0) {
        head_bytes += MODEL_LIANMENT - head_bytes % MODEL_LIANMENT;
        rcount = fread(model_alignment, sizeof(char), MODEL_LIANMENT - head_bytes % MODEL_LIANMENT, model_file);
    }
    size_t model_size = file_size - head_bytes;

    printf("model_size: %ld bytes, via %f KB, via %f MB, via %f GB\n", 
            model_size, (float)model_size / 1024, (float)model_size / 1024 / 1024, (float)model_size / 1024 / 1024 / 1024);

    // fclose(model_file);
    // int fd = open(checkpoint_path, O_RDONLY);
    // if (fd == -1) { fprintf(stderr, "open failed!\n"); exit(EXIT_FAILURE); }
    // void *data;
    // data = mmap(NULL, file_size, PROT_READ, MAP_SHARED | MAP_FILE, fd, 0);
    // if (data == MAP_FAILED) { fprintf(stderr, "mmap failed!\n"); exit(EXIT_FAILURE); }
    // char *host_memory = (char*)data + sizeof(int) + sizeof(Qwen2Config);

    // hipHostMalloc((void **)&data, model_size);


    char *host_memory = (char*)malloc(model_size);
    size_t chunck_size = 1024 * 1024 * 512; 
    size_t n_chuncks = model_size / chunck_size;
    size_t tail_size = model_size % chunck_size;

    printf("loading model from disk to host memory chuncks: %ld ......\n", n_chuncks);
    for (size_t i = 0; i < n_chuncks; i++) {
        rcount = fread(host_memory + i * chunck_size, sizeof(char), chunck_size, model_file);
        if (rcount != chunck_size) {
            fprintf(stderr, "Bad read model from model file %s\n", checkpoint_path);
            exit(1);
        }
    #ifdef WEIGHTS_DEBUG
        printf("n_chuncks:%lu the %lu chuncks\n", n_chuncks, i);
    #endif
    }

    if (tail_size > 0) {
        rcount = fread(host_memory + n_chuncks * chunck_size, sizeof(char), tail_size, model_file);
        if (rcount != tail_size) {
            fprintf(stderr, "Bad read model from model file %s\n", checkpoint_path);
            exit(1);
        }
    }


    // // https://people.csail.mit.edu/xchen/gpu-programming/Lecture14-stream.pdf
    // char *host_memory;
    // hipHostAlloc((void **)&host_memory, model_size, hipHostMallocDefault);
    // rcount = fread(host_memory, sizeof(char), model_size, model_file);
    // if (rcount != model_size) {
    //     fprintf(stderr, "Bad read model from model file %s\n", checkpoint_path);
    //     exit(1);
    // }

    void *device_memory;
    hipError_t err;
    printf("loading model from host memory to device memory......\n");
    hipMalloc((void**)&device_memory, model_size);
    ld_infer::cuda::CudaCheckError();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // printf("%s\n", hipGetErrorName(err));
    hipMemcpy(device_memory, host_memory, model_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("milliseconds: %.3f ms \n", milliseconds);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed %ld\n", model_size);
        exit(-1);
    }
    memory_map_weights(&model->weights, &model->config, (char*)device_memory);
    
    // free(host_memory);
}

typedef struct {

} Context;

__device__ bool thread0() {
    return (!threadIdx.x && !threadIdx.y && !threadIdx.z) && (!blockIdx.x && !blockIdx.y && !blockIdx.z);
}


void* qwen2_forward(Context *ctx, hipblasHandle_t *handle, Qwen2* qwen2, int *token, int batch, int pos) {
    Qwen2Config *p = &qwen2->config;
    Qwen2Weights *w = &qwen2->weights;
    RunState* s = &qwen2->state;

    s->flops = 0;
    s->flops_sfu = 0;
    int max_seq_len = s->max_seq_len;
    // float *x = s->x;

    int hidden_size = p->hidden_size;
    int intermediate_size = p->intermediate_size;
    // int max_position_embeddings = p->max_position_embeddings;
    // int max_window_layers = p->max_window_layers;
    int num_attention_heads = p->num_attention_heads;
    int num_hidden_layers = p->num_hidden_layers;
    int num_key_value_heads = p->num_key_value_heads;
    float rms_norm_eps = p->rms_norm_eps;
    float rope_theta = p->rope_theta;
    // int sliding_window = p->sliding_window;
    int vocab_size = p->vocab_size;

    int num_heads = num_attention_heads;
    int head_dim = hidden_size / num_heads;
    
    ld_infer::cuda::embedding::embedding_fwd_launch<half>(s->half_x, w->embed_tokens, token, batch, hidden_size);
    ld_infer::cuda::CudaCheckError();

    // for(int l = 0; l < 1; l++) {
    for(int l = 0; l < p->num_hidden_layers; l++) {
        // attn_norm
        ld_infer::cuda::rmsnorm_half::rmsnorm_fwd_launch(s->half_xb, s->half_x, w->input_layernorm + l*hidden_size, rms_norm_eps, batch, hidden_size);

        int offset_k = l * max_seq_len * batch * num_key_value_heads * head_dim 
                         + pos * batch * num_key_value_heads * head_dim;
        int offset_v = l * max_seq_len * batch * num_key_value_heads * head_dim 
                         + pos * batch * num_key_value_heads * head_dim;

        s->half_k = s->half_key_cache + offset_k;
        s->half_v = s->half_value_cache + offset_v;

        ld_infer::cuda::linear_half::linear_fwd_launch(s->handle, s->half_q, s->half_xb, w->q_proj_w + l * hidden_size * (num_heads * head_dim), w->q_proj_b + l * (num_heads * head_dim), batch, hidden_size, num_heads * head_dim);        
        ld_infer::cuda::linear_half::linear_fwd_launch(s->handle, s->half_k, s->half_xb, w->k_proj_w + l * hidden_size * (num_key_value_heads * head_dim), w->k_proj_b + l * (num_key_value_heads * head_dim), batch, hidden_size, num_key_value_heads * head_dim);
        ld_infer::cuda::linear_half::linear_fwd_launch(s->handle, s->half_v, s->half_xb, w->v_proj_w + l * hidden_size * (num_key_value_heads * head_dim), w->v_proj_b + l * (num_key_value_heads * head_dim), batch, hidden_size, num_key_value_heads * head_dim);

        ld_infer::cuda::rope_half::rope_launch(s->half_q, rope_theta, batch, num_heads, head_dim, pos);
        ld_infer::cuda::rope_half::rope_launch(s->half_k, rope_theta, batch, num_key_value_heads, head_dim, pos);

        ld_infer::cuda::flash_attention_half::flash_attention_half_fwd_launch(s->half_xb, s->half_q, s->half_key_cache, s->half_value_cache, 
                             batch, num_heads, num_key_value_heads, head_dim, num_heads, num_key_value_heads, max_seq_len, 
                             num_hidden_layers, l, pos);



        ld_infer::cuda::linear_half::linear_fwd_launch(s->handle, s->half_xb2, s->half_xb, w->o_proj + l * (num_heads * head_dim) * hidden_size, NULL, batch, num_heads * head_dim, hidden_size);

        ld_infer::cuda::residual_half::residual_fwd_launch(s->half_x, s->half_xb2, batch, hidden_size);

        // ffn_norm
        ld_infer::cuda::rmsnorm_half::rmsnorm_fwd_launch(s->half_xb, s->half_x, w->post_attention_layernorm + l*hidden_size, rms_norm_eps, batch, hidden_size);
        ld_infer::cuda::linear_half::linear_fwd_launch(s->handle, s->half_hb, s->half_xb, w->gate_proj + l*intermediate_size*hidden_size, NULL, batch, hidden_size, intermediate_size);
        ld_infer::cuda::linear_half::linear_fwd_launch(s->handle, s->half_hb2, s->half_xb, w->up_proj + l*intermediate_size*hidden_size, NULL, batch, hidden_size, intermediate_size);

        ld_infer::cuda::silu_half::silu_fwd_launch(s->half_hb, s->half_hb2, batch, intermediate_size);

        ld_infer::cuda::linear_half::linear_fwd_launch(s->handle, s->half_xb, s->half_hb, w->down_proj + l* hidden_size * intermediate_size, NULL, batch, intermediate_size, hidden_size);
                
        ld_infer::cuda::residual_half::residual_fwd_launch(s->half_x, s->half_xb, batch, hidden_size);

        // hipDeviceSynchronize();
    }
    
    ld_infer::cuda::rmsnorm_half::rmsnorm_fwd_launch(s->half_x, s->half_x, w->norm, rms_norm_eps, batch, hidden_size);
    ld_infer::cuda::linear_half::linear_fwd_launch1(s->handle, s->half_logits, s->half_x, w->lm_head, NULL, batch, hidden_size, vocab_size);

    return s->half_logits;
}


Qwen2 py_model;
hipblasHandle_t handle;

void c_init(int batch, int max_seq_len, const char *checkpoint_path) {
    printf("checkpoint_path: %s\n", checkpoint_path);
    if (checkpoint_path == NULL) {
        checkpoint_path = "qwen1.5-0.5B.bin";
    }
    // const char *checkpoint_path = "qwen1.5-0.5B.bin";
    qwen2_build_from_checkpoint(&py_model, checkpoint_path);
    py_model.state.batch = batch;
    py_model.state.max_seq_len = max_seq_len;

    hipblasCreate(&handle);
    py_model.state.handle = &handle;
    malloc_run_state(&py_model.state, &py_model.config);
    // hipblasDestroy(handle);
}

// void get_mod
int* c_qwen2_forward(int batch, int seq_len, int *data, int pos) {
    // printf("c_openelm_forward batch:%d, seq_len:%d, pos:%d\n", batch, seq_len, pos);
    RunState *s = &py_model.state;
    
    // int* prompt_tokens = data;
    // int start = 0;
    for (int i = 0; i < batch; i++) {
        // s->token[i] = data[i];
        hipMemcpy(s->token + i, data + i, sizeof(int), hipMemcpyHostToDevice);
    }
    
    Context ctx;
    qwen2_forward(&ctx, s->handle, &py_model, s->token, batch, pos);
    // hipDeviceSynchronize();
    ld_infer::cuda::argmax_half::argmax_fwd_launch(s->next, s->half_logits, s->batch, py_model.config.vocab_size);

    // if (pos == 10) {
    //     hipDeviceSynchronize();
    //     exit(1);
    // }

    for (int i = 0; i < s->batch; i++) {
        hipMemcpy(s->next_cpu + i, s->next + i, sizeof(int), hipMemcpyDeviceToHost);
    }
    
    // printf("pos:%d ", pos+1);
    // for (int i = 0; i < s->batch; i++) {
    //     printf("%d ", s->next_cpu[i]);
    // }
    // printf("\n");
    return s->next_cpu;
}
