/*
nvcc --shared -Xcompiler -fPIC -o qwen2.so -O3 qwen2.cu -lm -gencode arch=compute_86,code=sm_86
python run.py
*/


#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <math.h>
#include <fcntl.h>
#include <stdint.h>
#include <hip/hip_fp16.h>
#include <unistd.h>
#include <sys/mman.h>

extern "C" {
    void c_init(int batch, int max_seq_len, const char *checkpoint_path);
    int* c_qwen2_forward(int batch, int seq_len, int *data, int pos);
    // void c_generate(int batch, int seq_len, int *data, int steps);
    // void c_chat ();
}

constexpr int MODEL_LIANMENT{16};
constexpr int WARP_THREADS{32};
constexpr int WARPGROUP_THREADS{128};
constexpr int WARPGROUP_WARPS{4};

typedef struct {
    int hidden_size;
    int intermediate_size;
    int max_position_embeddings;
    int max_window_layers;
    int num_attention_heads;
    int num_hidden_layers;
    int num_key_value_heads;
    float rms_norm_eps;
    float rope_theta;
    int sliding_window;
    int vocab_size;
} Qwen2Config;

typedef struct {
    half *embed_tokens;    // model.embed_tokens.weight
    half *q_proj_w;        // model.layers.{i}.self_attn.q_proj.weight
    half *q_proj_b;        // model.layers.{i}.self_attn.q_proj.bias
    half *k_proj_w;        // model.layers.{i}.self_attn.k_proj.weight
    half *k_proj_b;        // model.layers.{i}.self_attn.k_proj.bias
    half *v_proj_w;        // model.layers.{i}.self_attn.v_proj.weight
    half *v_proj_b;        // model.layers.{i}.self_attn.v_proj.bias
    half *o_proj;          // model.layers.{i}.self_attn.o_proj.weight
    half *gate_proj;       // model.layers.{i}.mlp.gate_proj.weight
    half *up_proj;                    // model.layers.{i}.mlp.up_proj.weight
    half *down_proj;                  // model.layers.{i}.mlp.down_proj.weight
    half *input_layernorm;            // model.layers.{i}.input_layernorm.weight
    half *post_attention_layernorm;   // model.layers.{i}.post_attention_layernorm.weight
    half *norm;            // model.norm.weight
    half *lm_head;         // lm_head.weight
} Qwen2Weights;

typedef struct {
    float *x;
    float *xb;
    float *xb2;
    float *hb;
    float *hb2;
    float *q;
    float *k;
    float *v;
    float *key_cache;
    float *value_cache;
    float *att;
    float *logits;
    int *next;
    int *token;
    int *next_cpu;

    int batch;
    int max_seq_len;

    int flops;
    int flops_sfu;

    int num_parameters;
} RunState;

typedef struct {
    Qwen2Config config;
    RunState state;
    Qwen2Weights weights;
} Qwen2;

void malloc_run_state(RunState* s, Qwen2Config* p) {
    int seq_len = s->max_seq_len;
    int batch = s->batch;
    int hidden_size = p->hidden_size;
    int intermediate_size = p->intermediate_size;
    
    int num_heads = p->num_attention_heads;
    int head_dim = p->hidden_size / num_heads;
    int num_key_value_heads = p->num_key_value_heads;

    int num_hidden_layers = p->num_hidden_layers;

    unsigned long long run_cache = 0;

    hipMalloc((void**)&s->x, batch * hidden_size * sizeof(float));
    run_cache += batch * hidden_size * sizeof(float);
    hipMalloc((void**)&s->xb, batch * hidden_size * sizeof(float));
    run_cache += batch * hidden_size * sizeof(float);
    hipMalloc((void**)&s->xb2, batch * hidden_size * sizeof(float));
    run_cache += batch * hidden_size * sizeof(float);
    hipMalloc((void**)&s->hb, batch * intermediate_size * sizeof(float));
    run_cache += batch * intermediate_size * sizeof(float);
    hipMalloc((void**)&s->hb2, batch * intermediate_size * sizeof(float));
    run_cache += batch * intermediate_size * sizeof(float);
    hipMalloc((void**)&s->q, batch * hidden_size * sizeof(float));
    run_cache += batch * hidden_size * sizeof(float);
    hipMalloc((void**)&s->att, s->batch * num_heads * seq_len * sizeof(float));
    run_cache += s->batch * num_heads * seq_len * sizeof(float);
    unsigned long long kv_cache_size = batch * num_hidden_layers * seq_len * num_key_value_heads * head_dim * sizeof(float);
    hipMalloc((void**)&s->key_cache, kv_cache_size);
    run_cache += kv_cache_size;
    hipMalloc((void**)&s->value_cache, kv_cache_size);
    run_cache += kv_cache_size;
    printf("total kv cache size: %llu bytes, via %fKB, via %fMB, via %fGB\n", 2 * kv_cache_size, 
            (float)kv_cache_size  * 2.0 / 1024, (float)kv_cache_size  * 2.0 / 1024 / 1024, (float)kv_cache_size  * 2.0 / 1024 / 1024 / 1024);
    hipMalloc((void**)&s->logits, batch * p->vocab_size * sizeof(float));
    run_cache += batch * p->vocab_size * sizeof(float);
    hipMalloc((void**)&s->next, batch * sizeof(int));
    run_cache += batch * sizeof(int);
    hipMalloc((void**)&s->token, batch * sizeof(int));
    run_cache += batch * sizeof(int);
    printf("total run cache size: %llu bytes, via %fKB, via %fMB, via %fGB\n", run_cache, 
            (float)run_cache / 1024, (float)run_cache / 1024 / 1024, (float)run_cache / 1024 / 1024 / 1024);

    s->next_cpu = (int*)malloc(batch * sizeof(int));
}

void free_run_state(RunState* s) {
    hipFree(s->x);
    hipFree(s->xb);
    hipFree(s->xb2);
    hipFree(s->hb);
    hipFree(s->hb2);
    hipFree(s->q);
    hipFree(s->att);
    hipFree(s->key_cache);
    hipFree(s->value_cache);
    hipFree(s->logits);
    hipFree(s->next);
    hipFree(s->token);
    free(s->next_cpu);
}

void memory_map_weights(Qwen2Weights *w, Qwen2Config* p, char* ptr) {
    unsigned long long ll;
    unsigned long long ll_bytes;;
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->embed_tokens = (half*)ptr;
    ptr += ll_bytes;
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->q_proj_w = (half*)ptr;
    ptr += ll_bytes;
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->q_proj_b = (half*)ptr;
    ptr += ll_bytes;
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->k_proj_w = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->k_proj_b = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->v_proj_w = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->v_proj_b = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->o_proj = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->gate_proj = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->up_proj = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->down_proj = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->input_layernorm = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->post_attention_layernorm = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->norm = (half*)ptr;
    ptr += ll * sizeof(half);
    hipMemcpy(&ll, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    hipMemcpy(&ll_bytes, ptr, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    ptr += sizeof(unsigned long long);
    // printf("++++++++++++--------%llu\n", ll);
    // printf("++++++++++++--------%llu\n", ll_bytes);
    w->lm_head = (half*)ptr;
}

void qwen2_build_from_checkpoint(Qwen2 *model, const char* checkpoint_path) {
    FILE *model_file = fopen(checkpoint_path, "rb");
    if (model_file == NULL) {
        printf("Error opening model file %s\n", checkpoint_path);
    }
    char model_alignment[MODEL_LIANMENT];
    size_t file_size = 0;
    fseek(model_file, 0, SEEK_END);
    file_size = ftell(model_file);
    fseek(model_file, 0, SEEK_SET);
    printf("file_size is: %ld\n", file_size);
    
    int rcount = 0;
    int model_magic;
    rcount = fread(&model_magic, sizeof(int), 1, model_file);
    if (rcount != 1) {
        fprintf(stderr, "Bad read magic from model file %s\n", checkpoint_path);
        exit(1);
    }
    
    if (model_magic != 20240516) {
        fprintf(stderr, "Bad magic model file %s\n", checkpoint_path);
        exit(1);
    }
    printf("model magic is: %d\n", model_magic);

    rcount = fread(&model->config, sizeof(int), sizeof(model->config) / sizeof(int), model_file);
    if (rcount != sizeof(model->config) / sizeof(int)) {
        fprintf(stderr, "Bad read config from model file %s\n", checkpoint_path);
        exit(1);
    }
    printf("config hidden_size is: %d\n", model->config.hidden_size);
    printf("config intermediate_size is: %d\n", model->config.intermediate_size);
    printf("config max_position_embeddings is: %d\n", model->config.max_position_embeddings);
    printf("config max_window_layers is: %d\n", model->config.max_window_layers);
    printf("config num_attention_heads is: %d\n", model->config.num_attention_heads);
    printf("config num_hidden_layers is: %d\n", model->config.num_hidden_layers);
    printf("config num_key_value_heads is: %d\n", model->config.num_key_value_heads);
    printf("config rms_norm_eps is: %f\n", model->config.rms_norm_eps);
    printf("config rope_theta is: %f\n", model->config.rope_theta);
    printf("config sliding_window is: %d\n", model->config.sliding_window);
    printf("config vocab_size is: %d\n", model->config.vocab_size);

    size_t head_bytes = sizeof(model->config) + sizeof(int);
    if (head_bytes % MODEL_LIANMENT != 0) {
        head_bytes += MODEL_LIANMENT - head_bytes % MODEL_LIANMENT;
        rcount = fread(model_alignment, sizeof(char), MODEL_LIANMENT - head_bytes % MODEL_LIANMENT, model_file);
    }
    size_t model_size = file_size - head_bytes;
    printf("model_size: %ld bytes, via %f KB, via %f MB, via %f GB\n", 
            model_size, (float)model_size / 1024, (float)model_size / 1024 / 1024, (float)model_size / 1024 / 1024 / 1024);

    // fclose(model_file);
    // int fd = open(checkpoint_path, O_RDONLY);
    // if (fd == -1) { fprintf(stderr, "open failed!\n"); exit(EXIT_FAILURE); }
    // void *data;
    // data = mmap(NULL, file_size, PROT_READ, MAP_SHARED | MAP_FILE, fd, 0);
    // if (data == MAP_FAILED) { fprintf(stderr, "mmap failed!\n"); exit(EXIT_FAILURE); }
    // char *host_memory = (char*)data + sizeof(int) + sizeof(Qwen2Config);

    // cudaMallocHost((void **)&data, model_size);


    char *host_memory = (char*)malloc(model_size);
    size_t chunck_size = 1024 * 1024 * 512; 
    size_t n_chuncks = model_size / chunck_size;
    size_t tail_size = model_size % chunck_size;

    printf("loading model from disk to host memory......\n");
    for (size_t i = 0; i < n_chuncks; i++) {
        rcount = fread(host_memory + i * chunck_size, sizeof(char), chunck_size, model_file);
        if (rcount != chunck_size) {
            fprintf(stderr, "Bad read model from model file %s\n", checkpoint_path);
            exit(1);
        }
        printf("n_chuncks:%lu the %lu chuncks\n", n_chuncks, i);
    }

    if (tail_size > 0) {
        printf("tail_size:%lu \n", tail_size);
        rcount = fread(host_memory + n_chuncks * chunck_size, sizeof(char), tail_size, model_file);
        if (rcount != tail_size) {
            fprintf(stderr, "Bad read model from model file %s\n", checkpoint_path);
            exit(1);
        }
    }


    // // https://people.csail.mit.edu/xchen/gpu-programming/Lecture14-stream.pdf
    // char *host_memory;
    // cudaHostAlloc((void **)&host_memory, model_size, cudaHostAllocDefault);
    // rcount = fread(host_memory, sizeof(char), model_size, model_file);
    // if (rcount != model_size) {
    //     fprintf(stderr, "Bad read model from model file %s\n", checkpoint_path);
    //     exit(1);
    // }

    void *device_memory;
    hipError_t err;
    printf("loading model from host memory to device memory......\n");
    hipMalloc((void**)&device_memory, model_size);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed %ld %s\n", model_size, hipGetErrorName(err));
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // printf("%s\n", cudaGetErrorName(err));
    hipMemcpy(device_memory, host_memory, model_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("milliseconds: %.3f ms \n", milliseconds);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed %ld\n", model_size);
        exit(-1);
    }
    memory_map_weights(&model->weights, &model->config, (char*)device_memory);
    
    // free(host_memory);
}

typedef struct {

} Context;

__device__ bool thread0() {
    return (!threadIdx.x && !threadIdx.y && !threadIdx.z) && (!blockIdx.x && !blockIdx.y && !blockIdx.z);
}

__global__ 
void get_content_row(float *x, half* embed_tokens, int *token, int batch, int dim) {
    int bidx = blockIdx.x; // batch
    int bidy = blockIdx.y; // dim = 
    int tidx = threadIdx.x;
    int offset_x = bidx * dim + bidy * blockDim.x + tidx;
    int offset_t = bidy * blockDim.x + tidx;
    x[offset_x] = *(embed_tokens + token[bidx] * dim + offset_t);

    // if (thread0()) {
    //     printf("[");
    //     for (int b = 0; b < batch; b++) {
    //         int offset_x = b * dim;
    //         printf("[");
    //         for (int i = 0; i<dim; i++) {
    //             printf("%f, ", x[offset_x + i]);
    //         }
    //         printf("],\n");
    //     }
    //     printf("]\n");
    // }
}

// https://arxiv.org/pdf/1910.07467
__global__
void rmsnorm_forward(float* o, float* x, half *weight, float rms_norm_eps, int batch, int dim) {
    int bidx = blockIdx.x; // batch
    int bidy = blockIdx.y;
    int tid = threadIdx.x; // thread id
    int lid = tid % 32; // lane id
    
    float ss = 0.0f;
    int offset = bidx * dim;
    #pragma unroll
    for (int i = lid; i < dim; i += WARP_THREADS) {
        ss += x[offset + i] * x[offset + i];
    }
    __syncwarp();

    #pragma unroll
    for (int mask = 32 / 2; mask > 0; mask /= 2) {
        ss += __shfl_xor_sync(uint32_t(-1), ss, mask);
        __syncwarp();
    }

    ss /= dim;
    ss += rms_norm_eps;
    ss = rsqrtf(ss);

    int offset_x = bidx * dim + bidy * blockDim.x + tid;
    int offset_w = bidy * blockDim.x + tid;
    int offset_o = bidx * dim + bidy * blockDim.x + tid;
    o[offset_o] = x[offset_x] * ss * __half2float(weight[offset_w]);

    // if (thread0()) {
    //     printf("rmsnorm:\n");
    //     for (int b = 0; b < batch; b++) {
    //         int offset = b * dim;
    //         printf("[");
    //         for (int d = 0; d < dim; d++) {
    //              printf("%f, ", o[offset + d]);
    //         }
    //         printf("],\n");
    //     }
    // }
}

// https://pytorch.org/docs/stable/generated/torch.nn.Linear.html
__global__
void linear_forward(float* output, float* input, half *weight, half* bias, int batch, int in_features, int out_features) {
    int bidy = blockIdx.y;
    int tid = threadIdx.x;
    int laneid = tid % WARP_THREADS;
    int num_per_thread = in_features / WARP_THREADS;
    // int col = laneid * num_per_thread;
    int row = bidy * WARPGROUP_WARPS + threadIdx.y;

    float4* input4 = reinterpret_cast<float4*>(input);
    float4* weight4 = reinterpret_cast<float4*>(weight + row * in_features);

    float ss = 0.0f;

    for (int i = 0; i < (num_per_thread >> 3); i++) {
        int col_i = laneid + i * WARP_THREADS;
        if (col_i < in_features >> 3) {
        float4 a = input4[2 * col_i];
        float4 b = weight4[col_i];
        half2* b_0 = reinterpret_cast<half2*>(&b.x);
        half2* b_1 = reinterpret_cast<half2*>(&b.y);
        half2* b_2 = reinterpret_cast<half2*>(&b.z);
        half2* b_3 = reinterpret_cast<half2*>(&b.w);

        ss += a.x * __half2float(b_0->x);
        ss += a.y * __half2float(b_0->y);
        ss += a.z * __half2float(b_1->x);
        ss += a.w * __half2float(b_1->y);
        float4 c = input4[2 * col_i + 1];
        ss += c.x * __half2float(b_2->x);
        ss += c.y * __half2float(b_2->y);
        ss += c.z * __half2float(b_3->x);
        ss += c.w * __half2float(b_3->y);
        }
    }

    ss += __shfl_down_sync(0xffffffff, ss, 16);
    ss += __shfl_down_sync(0xffffffff, ss, 8);
    ss += __shfl_down_sync(0xffffffff, ss, 4);
    ss += __shfl_down_sync(0xffffffff, ss, 2);
    ss += __shfl_down_sync(0xffffffff, ss, 1);

    if (laneid == 0) {
        output[row] = ss;
    }
                
    if (bias != NULL) {
        output[row] += __half2float(bias[row]);
    } 

    // if (thread0()) {
    //     printf("linear:\n");
    //     for (int b = 0; b < batch; b++) {
    //         printf("[");
    //         for (int i = 0; i < out_features; i++) {
    //             printf("%f, ", output[b * out_features + i]);
    //         }
    //         printf("]\n");
    //     }
    //     printf("]\n");
    // }
}

__global__ 
void rope_forward(float *q, float rope_freq_constant, int batch, int q_heads, int head_dim, int pos) {

    int b = blockIdx.x;
    int h = blockIdx.y;
    int tid = threadIdx.x;
    // int lid = tid % WARP_THREADS;

    int offset = b * q_heads * head_dim + h * head_dim;

    for (int hd = tid; hd < head_dim / 2; hd += WARPGROUP_THREADS) {
        float v0 = q[offset + hd];
        float v1 = q[offset + hd + head_dim / 2];

        float freq = 1.0f / powf(rope_freq_constant, ((float)(2 * hd) / head_dim));
        // printf("sl=%d %d=%f ", sl, hd, sl * freq);
        float cos_val = cosf(pos * freq);
        float sin_val = sinf(pos * freq);
        // printf("sl=%d %d=%f ", sl, hd, sin_val);
        q[offset + hd] = v0 * cos_val - v1 * sin_val;
        q[offset + head_dim / 2 + hd] = v1 * cos_val + v0 * sin_val;
    }

    // if (thread0()) {
    //     printf("rope: \n");
    //     for (int b = 0; b < batch; b++) {
    //         printf("[");
    //         for (int h = 0; h < q_heads; h++) {
    //             printf("[");    
    //             int offset = b * q_heads * head_dim + h * head_dim;
    //             for (int hd = 0; hd < head_dim; hd++) {     
    //                 printf("%f,", q[offset + hd]);
    //             }
    //             printf("],\n");
    //         }
    //         printf("],\n");
    //     }
    // }
}

// https://courses.cs.washington.edu/courses/cse599m/23sp/notes/flashattn.pdf
__global__
void group_flash_attention_forward(float* output, float *q, float *key_cache, float *value_cache, float *att,
                             int batch, int q_heads, int k_heads, int head_dim, int max_q_heads, int max_kv_heads, int max_seq_len, 
                             int num_transformer_layers, int layer_idx, int pos) {
    int num_groups = q_heads / k_heads;
    int b = blockIdx.x;
    int h = blockIdx.y;
    int tid = threadIdx.x;
    int lid = tid % WARP_THREADS;
    int kNThreads = blockDim.x;
    extern __shared__ float smem_[];

    float *d = smem_;
    float *m = smem_ + 1;
    float *o = smem_ + 2;
    
    *d = 1.0f;

    int offset_q = b * q_heads * head_dim + h * head_dim;
    int offset_k = b * num_transformer_layers * max_seq_len * max_kv_heads * head_dim 
                 + layer_idx * max_seq_len * max_kv_heads * head_dim 
                 + 0 * max_kv_heads * head_dim
                 + (h / num_groups)  * head_dim;
        
    float score = 0.0f;
    for (int i = lid; i < head_dim; i += WARP_THREADS){
        score += q[offset_q + i] * key_cache[offset_k + i];
    }

    __syncwarp();

    #pragma unroll
    for (int mask = 32 / 2; mask > 0; mask /= 2) {
        score += __shfl_down_sync(uint32_t(-1), score, mask);
        // __syncwarp();
    }
    __syncwarp();
    if (lid == 0) {
        score /= sqrtf((float)head_dim);
        *m = score;
    }
    

    int offset_o = b * q_heads * head_dim + h * head_dim;
    for (int lv = tid; lv < head_dim; lv += WARPGROUP_THREADS){
        int offset_v = b * num_transformer_layers * max_seq_len * max_kv_heads * head_dim 
                     + layer_idx * max_seq_len * max_kv_heads * head_dim 
                     + 0 * max_kv_heads * head_dim
                     + (h / num_groups) * head_dim;
        o[lv] = value_cache[offset_v + lv];
        output[offset_o + lv] = o[lv];
    }

    
    
    // flash attention
    float m_i1 = 0.0f;
    float m_i = 0.0f;
    float d_i1 = 0.0f;
    float d_i = 0.0f;
    float o_i1 = 0.0f;
    float o_i = 0.0f;

    __syncthreads();
    for (int lk = 1; lk < pos + 1; lk++) {
        int offset_k = b * num_transformer_layers * max_seq_len * max_kv_heads * head_dim 
                         + layer_idx * max_seq_len * max_kv_heads * head_dim 
                         + lk * max_kv_heads * head_dim
                         + (h / num_groups)  * head_dim;
        
        // score = 0.0f;
        // for (int i = 0; i < head_dim; i++) {
        //     score += q[offset_q + i] * key_cache[offset_k + i];
        // }
        score = 0.0f;
        for (int i = lid; i < head_dim; i += WARP_THREADS){
            score += q[offset_q + i] * key_cache[offset_k + i];
        }

        __syncwarp();

        #pragma unroll
        for (int mask = 32 / 2; mask > 0; mask /= 2) {
            score += __shfl_xor_sync(uint32_t(-1), score, mask);
            // __syncwarp();
        }

        score /= sqrtf((float)head_dim);

        // att[offset_att + lk] = score;
        m_i1 = *m;
        m_i = m_i1;
        if (score > m_i1) {
            m_i = score;
        }

        d_i1 = *d;

        d_i = d_i1 * __expf(m_i1 - m_i) + __expf(score - m_i);

        __syncthreads();
        for (int lv = tid; lv < head_dim; lv += kNThreads){
            o_i1 = o[lv];
            int offset_v = b * num_transformer_layers * max_seq_len * max_kv_heads * head_dim 
                         + layer_idx * max_seq_len * max_kv_heads * head_dim 
                         + lk * max_kv_heads * head_dim
                         + (h / num_groups) * head_dim;
            o_i = o_i1 * (d_i1 * __expf(m_i1 - m_i) / d_i) + __expf(score - m_i) / d_i * value_cache[offset_v + lv];
            o[lv] = o_i;
            output[offset_o + lv] = o_i;
        }

        *d = d_i;
        *m = m_i;
    }

    // if (thread0()) {
    //     printf("group_attention:\n");
    //     for (int b = 0; b < batch; b++) {
    //         printf("[");
    //         for (int d = 0; d < q_heads * head_dim; d++) {
    //             int offset = b * q_heads * head_dim;
    //                 printf("%f, ",output[offset + d]);
    //         }
    //         printf("],\n");
    //     }
    // }
}

__global__
void residual_forward(float *x, float *xb, int batch, int dim) {
    int b = blockIdx.x;
    int bidy = blockIdx.y;
    int tid = threadIdx.x;
    int kNThreads = blockDim.x;
    int offset = b * dim + bidy * kNThreads + tid;

    x[offset] += xb[offset];

    // if (thread0()) {
    //     printf("residual:\n");
    //     for (int b = 0; b < batch; b++) {
    //         printf("[");
    //         for (int i = 0; i < dim; i++) {
    //             int offset_x = b * dim + i;
    //             printf("%f, ", x[offset_x]);
    //         }
    //         printf("]\n");
    //     }
    // }
}

// https://pytorch.org/docs/stable/generated/torch.nn.SiLU.html
__global__
void silu_forward(float *hb, float* hb2, int batch, int intermediate_dim) {

    int b = blockIdx.x;
    int bidy = blockIdx.y;
    int tid = threadIdx.x;

    int offset = b * intermediate_dim + bidy * WARPGROUP_THREADS + tid;

    float val = hb[offset];
    val *= 1.0f / (1.0f + __expf(-val));
    val *= hb2[offset];
    hb[offset] = val;

    // if (thread0()) {
    //     printf("silu:\n");
    //     for (int b = 0; b < batch; b++) {
    //         printf("[");
    //         for (int i = 0; i < intermediate_dim; i++) {
    //             printf("%f, ", hb[b * intermediate_dim + i]);
    //         }
    //         printf("]\n");
    //     }
    // }
}

__global__
void logits_forward(float* output, float* input, half *weight, half* bias, int batch, int in_features, int out_features) {
    int b = blockIdx.x;
    int bidy = blockIdx.y;
    int tid = threadIdx.x;
    int kNThreads = blockDim.x;

    int out = bidy * kNThreads + tid;
    int offset_out = b * out_features + out;
    int offset_bias = out;
    float value = 0.0f;
    for (int in = 0; in < in_features; in++) {
        int offset_in = b * in_features + in;
        int offset_weight = out * in_features + in;
        value += input[offset_in] * __half2float(weight[offset_weight]);
    }
    output[offset_out] = value;
    if (bias != NULL) {
        output[offset_out] += __half2float(bias[offset_bias]);
    } 

    // if (thread0()) {
    //     printf("logits: \n");
    //     for (int b = 0; b < batch; b++) {
    //         printf("[");
    //         for (int i = 0; i < out_features; i++) {
    //             printf("%f, ", output[b * out_features + i]);
    //         }
    //         printf("]\n");
    //     }
    // }
}

__global__
void argmax_forward(int* output, float* input, int batch, int dim) {
    int b = blockIdx.x;
    int tid = threadIdx.x;
    int lid = tid % 32; // lane id

    int offset = b * dim;

    int max_i = lid;
    float max_val = input[offset + max_i];
    
    for (int i = lid; i < dim; i += WARP_THREADS) { 
        if (input[offset + i] > max_val) {
            max_val = input[offset + i];
            max_i = i;
        }
    }

    __syncwarp();

    #pragma unroll
    for (int mask = 32 / 2; mask > 0; mask /= 2) {
        int shfl_i = __shfl_xor_sync(uint32_t(-1), max_i, mask);
        if (input[offset + shfl_i] > max_val) {
            max_val = input[offset + shfl_i];
            max_i = shfl_i;
        }
        __syncwarp();
    }
    
    output[b] = max_i;

    // if (thread0()) {
    //     printf("argmax:\n");
    //     printf("[");
    //     for (int b = 0; b < batch; b++) {
    //         printf("%d, ", output[b]);
    //     }
    //     printf("]\n");
    // }
}

void* qwen2_forward(Context *ctx, Qwen2* qwen2, int *token, int batch, int pos) {
    Qwen2Config *p = &qwen2->config;
    Qwen2Weights *w = &qwen2->weights;
    RunState* s = &qwen2->state;

    s->flops = 0;
    s->flops_sfu = 0;
    int max_seq_len = s->max_seq_len;
    float *x = s->x;

    int hidden_size = p->hidden_size;
    int intermediate_size = p->intermediate_size;
    // int max_position_embeddings = p->max_position_embeddings;
    // int max_window_layers = p->max_window_layers;
    int num_attention_heads = p->num_attention_heads;
    int num_hidden_layers = p->num_hidden_layers;
    int num_key_value_heads = p->num_key_value_heads;
    float rms_norm_eps = p->rms_norm_eps;
    float rope_theta = p->rope_theta;
    // int sliding_window = p->sliding_window;
    int vocab_size = p->vocab_size;

    int num_heads = num_attention_heads;
    int head_dim = hidden_size / num_heads;
    
    // printf("qwen2_forward pos:%d, batch:%d, hidden_size:%d \n", pos, batch, hidden_size);
    hipError_t err;
    
    get_content_row<<<dim3(batch, hidden_size/WARPGROUP_THREADS), WARPGROUP_THREADS>>>(x, w->embed_tokens, token, batch, hidden_size);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("%s\n", hipGetErrorName(err));
    }

    hipDeviceSynchronize();
    dim3 linear_grid_dim;
    dim3 linear_block_dim(WARP_THREADS, WARPGROUP_WARPS);
    // for(int l = 0; l < 1; l++) {
    for(int l = 0; l < p->num_hidden_layers; l++) {
        // attn_norm
        rmsnorm_forward<<<dim3(batch, hidden_size / WARPGROUP_THREADS), WARPGROUP_THREADS>>>(s->xb, s->x, w->input_layernorm + l*hidden_size, rms_norm_eps, batch, hidden_size);

        int offset_k = l * max_seq_len * num_key_value_heads * head_dim 
                         + pos * num_key_value_heads * head_dim;
        int offset_v = l * max_seq_len * num_key_value_heads * head_dim 
                         + pos * num_key_value_heads * head_dim;
        s->k = s->key_cache + offset_k;
        s->v = s->value_cache + offset_v;

        // batch * p->num_hidden_layers * seq_len * num_heads * head_dim
        linear_grid_dim = dim3(batch, num_heads * head_dim / WARPGROUP_WARPS);
        linear_forward<<<linear_grid_dim, linear_block_dim>>>(s->q, s->xb, w->q_proj_w + l * hidden_size * (num_heads * head_dim), w->q_proj_b + l * (num_heads * head_dim), batch, hidden_size, num_heads * head_dim);
        // cudaDeviceSynchronize();
        // exit(-1);
        linear_grid_dim = dim3(batch, num_key_value_heads * head_dim / WARPGROUP_WARPS);
        linear_forward<<<linear_grid_dim, linear_block_dim>>>(s->k, s->xb, w->k_proj_w + l * hidden_size * (num_key_value_heads * head_dim), w->k_proj_b + l * (num_key_value_heads * head_dim), batch, hidden_size, num_key_value_heads * head_dim);
        linear_grid_dim = dim3(batch, num_key_value_heads * head_dim / WARPGROUP_WARPS);
        linear_forward<<<linear_grid_dim, linear_block_dim>>>(s->v, s->xb, w->v_proj_w + l * hidden_size * (num_key_value_heads * head_dim), w->v_proj_b + l * (num_key_value_heads * head_dim), batch, hidden_size, num_key_value_heads * head_dim);
        
        rope_forward<<<dim3(batch, num_heads), WARPGROUP_THREADS>>>(s->q, rope_theta, batch, num_heads, head_dim, pos);

        rope_forward<<<dim3(batch, num_key_value_heads), WARPGROUP_THREADS>>>(s->k, rope_theta, batch, num_heads, head_dim, pos);

        // group attention
        group_flash_attention_forward<<<dim3(batch, num_heads), WARPGROUP_THREADS>>>(s->xb, s->q, s->key_cache, s->value_cache, s->att,
                             batch, num_heads, num_key_value_heads, head_dim, num_heads, num_key_value_heads, max_seq_len, 
                             num_hidden_layers, l, pos);

        linear_grid_dim = dim3(batch, hidden_size / WARPGROUP_WARPS);
        linear_forward<<<linear_grid_dim, linear_block_dim>>>(s->xb2, s->xb, w->o_proj + l * (num_heads * head_dim) * hidden_size, NULL, batch, num_heads * head_dim, hidden_size);

        residual_forward<<<dim3(batch, hidden_size / WARPGROUP_THREADS), WARPGROUP_THREADS>>>(s->x, s->xb2, batch, hidden_size);

        // ffn_norm
        rmsnorm_forward<<<dim3(batch, hidden_size / WARPGROUP_THREADS), WARPGROUP_THREADS>>>(s->xb, s->x, w->post_attention_layernorm + l*hidden_size, rms_norm_eps, batch, hidden_size);

        linear_grid_dim = dim3(batch, intermediate_size / WARPGROUP_WARPS);
        linear_forward<<<linear_grid_dim, linear_block_dim>>>(s->hb, s->xb, w->gate_proj + l*intermediate_size*hidden_size, NULL, batch, hidden_size, intermediate_size);
        linear_grid_dim = dim3(batch, intermediate_size / WARPGROUP_WARPS);
        linear_forward<<<linear_grid_dim, linear_block_dim>>>(s->hb2, s->xb, w->up_proj + l*intermediate_size*hidden_size, NULL, batch, hidden_size, intermediate_size);
  
        silu_forward<<<dim3(batch, intermediate_size / WARPGROUP_THREADS), WARPGROUP_THREADS>>>(s->hb, s->hb2, batch, intermediate_size);

        linear_grid_dim = dim3(batch, hidden_size / WARPGROUP_WARPS);
        linear_forward<<<linear_grid_dim, linear_block_dim>>>(s->xb, s->hb, w->down_proj + l* hidden_size * intermediate_size, NULL, batch, intermediate_size, hidden_size);

        residual_forward<<<dim3(batch, hidden_size / WARPGROUP_THREADS), WARPGROUP_THREADS>>>(s->x, s->xb, batch, hidden_size);

        // cudaDeviceSynchronize();
    }

    rmsnorm_forward<<<dim3(batch, hidden_size / WARPGROUP_THREADS), WARPGROUP_THREADS>>>(s->x, s->x, w->norm, rms_norm_eps, batch, hidden_size);
    
    logits_forward<<<dim3(batch, vocab_size / WARPGROUP_THREADS), WARPGROUP_THREADS>>>(s->logits, s->x, w->lm_head, NULL, batch, hidden_size, vocab_size);

    return s->logits;
}


Qwen2 py_model;

void c_init(int batch, int max_seq_len, const char *checkpoint_path) {
    printf("checkpoint_path: %s\n", checkpoint_path);
    if (checkpoint_path == NULL) {
        checkpoint_path = "qwen1.5-0.5B.bin";
    }
    // const char *checkpoint_path = "qwen1.5-0.5B.bin";
    qwen2_build_from_checkpoint(&py_model, checkpoint_path);
    py_model.state.batch = batch;
    py_model.state.max_seq_len = max_seq_len;
    malloc_run_state(&py_model.state, &py_model.config);
}

// void get_mod
int* c_qwen2_forward(int batch, int seq_len, int *data, int pos) {
    // printf("c_openelm_forward batch:%d, seq_len:%d, pos:%d\n", batch, seq_len, pos);
    RunState *s = &py_model.state;
    
    // int* prompt_tokens = data;
    // int start = 0;
    for (int i = 0; i < batch; i++) {
        // s->token[i] = data[i];
        hipMemcpy(s->token + i, data + i, sizeof(int), hipMemcpyHostToDevice);
    }
    
    Context ctx;
    qwen2_forward(&ctx, &py_model, s->token, batch, pos);
    // cudaDeviceSynchronize();
    argmax_forward<<<s->batch, WARPGROUP_THREADS>>>(s->next, s->logits, s->batch, py_model.config.vocab_size);
    hipDeviceSynchronize();

    for (int i = 0; i < s->batch; i++) {
        hipMemcpy(s->next_cpu + i, s->next + i, sizeof(int), hipMemcpyDeviceToHost);
    }
    
    // printf("pos:%d ", pos+1);
    // for (int i = 0; i < s->batch; i++) {
    //     printf("%d ", s->next_cpu[i]);
    // }
    // printf("\n");
    return s->next_cpu;
}
