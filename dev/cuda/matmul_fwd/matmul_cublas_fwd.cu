#include "hipblas.h"
#include <stdio.h>
template <typename H, typename F>
void launch_simple_gemm_tt(size_t m, size_t n, size_t k, F const* alpha,
                           H const* A, size_t lda, H const* B, size_t ldb,
                           F const* beta, F* C, size_t ldc,
                           hipStream_t stream) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t status = cublasSgemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                          m, n, k,
                                          alpha, 
                                          A, HIP_R_16F, k, 
                                          B, HIP_R_16F, k, 
                                          beta, 
                                          C, HIP_R_32F, m);
    hipDeviceSynchronize();
    hipblasDestroy(handle);
}

#include "matmul_forward_harness.impl"