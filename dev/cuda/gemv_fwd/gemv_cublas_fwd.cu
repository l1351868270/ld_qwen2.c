#include "hipblas.h"
#include <stdio.h>
namespace gemv {
template <typename H, typename F>
void launch_gemm(size_t m, size_t n, size_t k, F const* alpha,
                           H const* A, size_t lda, H const* B, size_t ldb,
                           F const* beta, F* C, size_t ldc,
                           hipStream_t stream) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t status = cublasSgemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                                          m, n, k,
                                          alpha, 
                                          A, HIP_R_16F, k, 
                                          B, HIP_R_16F, k, 
                                          beta, 
                                          C, HIP_R_32F, m);
    hipDeviceSynchronize();
    hipblasDestroy(handle);
}
} // namespace gemv
#include "gemv_fwd_harness.impl"